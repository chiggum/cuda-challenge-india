#include "hip/hip_runtime.h"
/*
 * CUDA CHALLENGE INDIA
 * 
 * Team Name: Flash
 * Team members: Dhruv Kohli, Vedant Kohli
 *
 * Libraries used: thrust
 *
 * Control Flow:
 * main -> processMap -> saturateMap -> saturateMapParallel ->
 * binarizeMap -> calcThreshold -> binarizeMapParallel -> 
 * performCCLnPrintNCC -> initLabels -> scanning -> analysis ->
 * printNCC -> computeNCC
 *
 * References:
 * Connected Component Labelling:
 * 1. K. Hawick, A. Leist, D. Playne, "Parallel graph component labelling with GPUs
 * and CUDA", Parallel Computing 36 (12) (2010) 655�678.
 * 2. O. Kalentev, A. Rai, S. Kemnitz, and R. Schneider, "Connected component labeling 
 * on a 2D grid using CUDA," J. Parallel Distributed Computing, pp. 615-620, 2011.
 * General:
 * 3. NVIDIA, Cuda programming guide 6.5.
 * Warp-Aggregated Atomics:
 * 4. CUDA Pro Tip: Optimized Filtering with Warp-Aggregated Atomics.
 * 
 * This code makes use of Label equivalence algorithm described in [2] for CCL.
 */

#include <stdlib.h>	//exit, malloc
#include "input_large.h"	//getinput
#include <thrust/device_vector.h>	//thrust::reduce
#include <stdio.h>	//printf

typedef unsigned int uint;

#define BLOCKSIZE 256
#define WARP_SZ 32
#define cudaMemcpyHTD(dest, src, nBytes) hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice)
#define cudaMemcpyDTH(dest, src, nBytes) hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost)

//process map
void processMap(uint*, uint, uint);

//finding hills and dales and replacing by mean and median respectively
void saturateMap(uint*, uint, uint);
__global__ void saturateMapParallel(uint*, uint*, bool*, uint, uint);

//binarize map
void binarizeMap(uint*, uint, uint);
int calcThreshold(uint*, uint, uint);
__global__ void binarizeMapParallel(uint*, uint, uint, uint);

//connected component labelling and finding number of connected components
void performCCLnPrintNCC(uint*, uint, uint);
__global__ void initLabels(uint*, uint*, uint, uint);
__global__ void scanning(uint*, bool*, uint, uint);
__global__ void analysis(uint*, uint, uint);
void printNCC(uint*, uint, uint);
__global__ void computeNCC(uint*, uint*, uint, uint);

//returns lane id of a thread in a warp
__device__ inline int lane_id();
//warp-aggregated atomic increment
__device__ void atomicAggInc(uint*);

int
main(int argc, char **argv) {
	uint *input = get_input();
	uint numMaps;
	uint numRows, numCols;

	if(input != NULL) {
		numMaps = input[0];
	} else {
		printf("Error: input is NULL!\n");
		std::exit(EXIT_FAILURE);
	}

	uint i = 0;
	uint offset = 1;
	while(i < numMaps) {
		numRows = input[offset];
		numCols = input[offset + 1];
		offset += 2;
		printf("MAP #%u: ", i+1);
		processMap(input + offset, numRows, numCols);
		offset += numRows * numCols;
		i++;
	}
	return 0;
}
/*
 * saturateMap description:
 * -Reuses buffer by swapping pointers. This prevents DTDcpy of the map.
 * -Calls SaturateMapParallel (one thread per cell)until all values converge or iterations exceed 
 *  max iterations.
 * -Once saturated, the map is binarized and CCL is performed over binarized map.
 */
void
saturateMap(uint *h_map, uint rows, uint cols) {
	//variable dec
	uint netCells = rows*cols, iter = 0;
	uint *d_mapOut, *d_mapIn;
	bool *d_notConverged, h_notConverged = true;//flags to check if values in map converged
	dim3 threadsPerBlock(BLOCKSIZE);
	dim3 numBlocks((netCells-1)/threadsPerBlock.x + 1); 

	//memory allocation on device and copy map data from host to device
	hipMalloc((void**)&d_mapIn, rows*cols*sizeof(uint));
	hipMalloc((void**)&d_mapOut, rows*cols*sizeof(uint));
	hipMalloc((void**)&d_notConverged, sizeof(bool));
	cudaMemcpyHTD(d_mapIn, h_map, rows*cols*sizeof(uint));

	//processing
	while(1) {
		cudaMemcpyHTD(d_notConverged, &h_notConverged, sizeof(bool));
		if(NUM_ITERATIONS != 0) {
			if(iter%2==0)
				saturateMapParallel<<<numBlocks, threadsPerBlock>>>(d_mapIn, d_mapOut, 
					d_notConverged, rows, cols);
			else
				saturateMapParallel<<<numBlocks, threadsPerBlock>>>(d_mapOut, d_mapIn, 
					d_notConverged, rows, cols);
		}
		cudaMemcpyDTH(&h_notConverged, d_notConverged, sizeof(bool));
		h_notConverged = !h_notConverged;
		++iter;
		//converged or max number of iterations reached
		if(iter >= NUM_ITERATIONS || !h_notConverged) {
			if(iter%2==1) {
				binarizeMap(d_mapOut, rows, cols);
				performCCLnPrintNCC(d_mapOut, rows, cols);
			} else {
				binarizeMap(d_mapIn, rows, cols);
				performCCLnPrintNCC(d_mapIn, rows, cols);
			}
			break;
		}
	}

	//free up the memory
	hipFree(d_notConverged);
	hipFree(d_mapIn);
	hipFree(d_mapOut);
}

/*
 * saturateMapParallel Description:
 * -Checks if value at idx of input map is a hill or Dale.
 * -If hill then put avg. of surrounding cells at idx of output map and raise flag not cvg.
 * -If dale then put median of surrounding cells at idx of outmap map and raise flag not cvg.
 * -Else put the same value i.e. value at idx in input map at idx of output map
 *
 * Other variants of this kernel which failed i.e. increased execution time:
 * -Shared Memory version increased execution time due to the synchronization barrier.
 *  It was observed that the net synchronization time was greater than the time to access 7
 *  extra global cells per thread in global version.
 * -Lookup table version which stores flag for each cell, whether the cell can be a hill or
 *  a dale in its lifetime. That means that if a cell and one of the surrounding cell has
 *  same value then that cell can never be a hill or a dale in its life time, this boolean
 *  flag is stored in the lookup table but that too took extra time due to more overhead in 
 *  global memory access than to return from the kernel if the cell has value equal to one of
 *  the corresponding cell.
 *
 */
__global__ void
saturateMapParallel(uint *inputMap, uint *outputMap, bool *notConverged, uint rows, uint cols) {
	uint idx = threadIdx.x + blockIdx.x*blockDim.x;
	uint i = idx/cols;
	uint j = idx%cols;
	if(i >= rows || j >= cols)
		return;
	uint cell = j+i*cols;
	uint focusElem = inputMap[cell];
	if(i == 0 || i == rows - 1 || j == 0 || j == cols - 1) {
		outputMap[cell] = focusElem;
		return;
	}
	uint localMap[8];
	uint cntU = 0, cntD = 0, cnt = 0, sum = 0;

	//checks if the cell is hill/dale/none
	for(uint l = i - 1; l <= i + 1; ++l) {
		for(uint k = j - 1; k <= j + 1; ++k) {
			if(!(l == i && k == j)) {
				localMap[cnt] = inputMap[k+l*cols];
				sum += localMap[cnt];
				if(localMap[cnt] > focusElem)
					++cntD;
				else if(localMap[cnt] < focusElem)
					++cntU;
				else {
					//cell is neither hill nor dale and can never be in its lifetime
					outputMap[cell]=focusElem;
					return;
				}
				++cnt;
			}
		}
	}
	if(cntU == 8) {//hill
		outputMap[cell]=sum/8;
		*notConverged = false;
	} else if(cntD == 8) {//dale
		//sorting to get first 5 terms in sorted array localMap
		for(int l = 0; l < 4; ++l) {
			for(int k = 0; k < 7 - l; ++k) {
				if(localMap[k] > localMap[k+1]) {
					uint temp = localMap[k];
					localMap[k] = localMap[k+1];
					localMap[k+1]=temp;
				}
			}
		}
		outputMap[cell] = (localMap[3]+localMap[4])/2;
		*notConverged = false;
	} else {//none
		outputMap[cell] = focusElem;
	}
		
}

/*
 * binarizeMap description:
 * -Obtains threshold and passes it to the binarizeMapParallel (with one thread per cell)
 *  which puts the binarized map in d_input.
 */
void
binarizeMap(uint *d_input, uint rows, uint cols) {
	uint netCells = rows*cols, threshold;
	dim3 threadsPerBlock(BLOCKSIZE);
	dim3 numBlocks((netCells-1)/threadsPerBlock.x + 1); 
	threshold = calcThreshold(d_input, rows, cols);
	binarizeMapParallel<<<numBlocks, threadsPerBlock>>>(d_input, rows, cols, threshold);
}

/*
 * calcThreshold description:
 * -Calculates the sum of the values in d_input map using thrust::reduce and then
 * divides the sum with rows*cols and finally returns this value as threshold.
 */
int
calcThreshold(uint *d_input, uint rows, uint cols) {
	uint sum, threshold;
	thrust::device_ptr<uint> dev_ptr(d_input);
	sum = thrust::reduce(dev_ptr, dev_ptr+rows*cols, (uint)0, thrust::plus<uint>());
	threshold = sum/(rows*cols);
	return threshold;
}

/*
 * binarizeMapParallel description:
 * -Checks whether the value of the input map represented by idx is less than threshold.If it
 *  is then assign 0 to that cell in the input map, else assign 1.
 */
__global__ void
binarizeMapParallel(uint *inputMap, uint rows, uint cols, uint threshold) {
	uint idx = threadIdx.x + blockIdx.x*blockDim.x;
	uint i = idx/cols;
	uint j = idx%cols;
	if(i >= rows || j >= cols)
		return;
	uint cell = j+i*cols;
	if(inputMap[cell] < threshold)
		inputMap[cell]=0;
	else
		inputMap[cell]=1;
}

/*
 * performCCLnPrintNCC description:
 * -Uses the algorithm mentioned in reference[2] to label the connected components of
 *  the input binary map.
 * -Then calls printNCC to calculate and print the no. of conn. comp. in the labelled map.
 */
void
performCCLnPrintNCC(uint *d_input, uint rows, uint cols) {
	//variable declaration
	uint netCells = rows*cols;
	uint *d_label; 
	bool h_notConverged = true, *d_notConverged;
	uint rowsPad = rows+2, colsPad = cols+2;
	dim3 threadsPerBlock(BLOCKSIZE);
	dim3 numBlocks((netCells-1)/threadsPerBlock.x + 1);

	//memory allocation
	hipMalloc((void**)&d_label, rowsPad*colsPad*sizeof(uint));
	hipMalloc((void**)&d_notConverged, sizeof(bool));

	//initializing labels
	initLabels<<<(rowsPad*colsPad-1)/threadsPerBlock.x+1, threadsPerBlock>>>(d_label, d_input, rowsPad, colsPad); 
	//computation
	while(h_notConverged) {
		cudaMemcpyHTD(d_notConverged, &h_notConverged, sizeof(bool));
		scanning<<<numBlocks, threadsPerBlock>>>(d_label, d_notConverged, rows, cols);
		cudaMemcpyDTH(&h_notConverged, d_notConverged, sizeof(bool));
		h_notConverged = !h_notConverged;
		if(h_notConverged) {
			analysis<<<numBlocks, threadsPerBlock>>>(d_label, rows, cols);
		}
	}
	printNCC(d_label, rows, cols);

	//free up the memory
	hipFree(d_notConverged);
	hipFree(d_label);
}

/*
 * initLabels description:
 * -If the cell corresponding to idx in the input map is on boundary then label of 
 *  that cell will be 0, else if it's value is zero then label of that cell is 0, 
 *  else if it's value is 1 then label of that cell will be the sequential index of that cell.
 */
__global__ void
initLabels(uint *label, uint *map, uint rows, uint cols) {
	uint idx = threadIdx.x + blockIdx.x*blockDim.x;
	uint i = idx/cols;
	uint j = idx%cols;
	if(i >= rows || j >= cols)
		return;
	uint cell_ = j+(i)*(cols);
	if(i == 0 || i == rows-1 || j == 0 || j == cols-1) {
		label[cell_]=0;
		return;
	}
	uint cell = j-1+(i-1)*(cols-2);
	label[cell_]=(cell_)*map[cell];
}

/*
 * scanning description:
 * -If the value of the label(LAB) represented by idx is less than minimum(MIN) value of the
 *  surrounding labels then assign label[LAB], the min of the label[LAB] and MIN and raise flag
 *  not converged else do nothing.
 */
__global__ void
scanning(uint *label, bool *notConverged, uint rows, uint cols) {
	uint idx = threadIdx.x + blockIdx.x*blockDim.x;
	uint i = idx/cols;
	uint j = idx%cols;
	if(i >= rows || j >= cols)
		return;
	uint cell_ = j+1+(i+1)*(cols+2);
	uint l = label[cell_];
	if(l == 0)
		return;
	uint lw = label[cell_-1];
	uint minl = (rows+2)*(cols+2) + 1;
	if(lw)minl=lw;
	uint le = label[cell_+1];
	if(le&&le<minl)minl=le;
	uint lwn = label[cell_-cols-3];
	if(lwn&&lwn<minl)minl=lwn;
	uint lws = label[cell_+cols+1];
	if(lws&&lws<minl)minl=lws;
	uint len = label[cell_-cols-1];
	if(len&&len<minl)minl=len;
	uint les = label[cell_+cols+3];
	if(les&&les<minl)minl=les;
	uint ln = label[cell_-cols-2];
	if(ln&&ln<minl)minl=ln;
	uint ls = label[cell_+cols+2];
	if(ls&&ls<minl)minl=ls;
	if(minl < l) {
		uint ll = label[l];
		if(minl<ll)
			label[l]=minl;
		else
			label[l]=ll;
		*notConverged=false;
	}
}

/*
 * analysis description:
 * -Starting with l equal to label corresponding to the idx and reference equal to
 *  label[l], we iterate by replacing l with label[reference] and then reference with 
 *  label[l] until reference becomes equal to l (let this value be VAL).
 * -Relabels the cell represented by idx with VAL.
 */
__global__ void
analysis(uint *label, uint rows, uint cols) {
	uint idx = threadIdx.x + blockIdx.x*blockDim.x;
	uint i = idx/cols;
	uint j = idx%cols;
	if(i >= rows || j >= cols)
		return;
	uint cell_ = j+1+(i+1)*(cols+2);
	uint l = label[cell_];
	if(l == 0)
		return;
	uint ref = label[l];
	while(ref!=l) {
		l=label[ref];
		ref=label[l];
	}
	label[cell_]=l;
}

/*
 * printNCC description:
 * -Calls computeNCC which computes the number of connected components in d_input
 *  which is nothing but the labels which match their sequential indices.
 * -And then it prints ncc(number of connected components).
 */
void
printNCC(uint *d_input, uint rows, uint cols) {
	uint netCells = rows*cols;
	dim3 threadsPerBlock(BLOCKSIZE);
	dim3 numBlocks((netCells-1)/threadsPerBlock.x + 1);
	uint *ncomp, ncc;
	//memory allocation and memset ncomp to zero
	hipMalloc((void**)&ncomp, sizeof(uint));
	hipMemset(ncomp, 0, sizeof(uint));
	//computes n0. of conn. comp. and stores in ncomp
	computeNCC<<<numBlocks, threadsPerBlock>>>(ncomp, d_input, rows, cols);
	//copy data(ncomp) from device to host
	cudaMemcpyDTH(&ncc, ncomp, sizeof(uint));
	//print ncc
	printf("%u\n", ncc);
	//free up the memory
	hipFree(ncomp);
}


/*
 * computeNCC description:
 * -If value of the label represented by idx is same as its index then do atomicAggInc 
 *  else return (label must be nonzero).
 */
__global__ void
computeNCC(uint *ncomp, uint *label, uint rows, uint cols) {
	uint idx = threadIdx.x + blockIdx.x*blockDim.x;
	uint i = idx/cols;
	uint j = idx%cols;
	if(i >= rows || j >= cols)
		return;
	uint cell_ = j+1+(i+1)*(cols+2);
	uint l = label[cell_];
	if(l == 0)
		return;
	if(l==cell_)
		atomicAggInc(ncomp);
}

/*
 * processMap description:
 * -Transfers control to saturateMap which initiates actual processing.
 */
void
processMap(uint *map, uint rows, uint cols) {
	saturateMap(map, rows, cols);
}

/*
 * returns lane id of a thread.
 */
__device__ inline int 
lane_id(void) {
	return threadIdx.x % WARP_SZ;
}

/*
 * warp-aggregated atomic increment.
 * Refer: http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
 * for explanation of Why AtomicAggInc is faster than thrust and atomicAdd.
 */
__device__ void 
atomicAggInc(uint *ctr) {
  int mask = __ballot(1);
  // select the leader
  int leader = __ffs(mask) - 1;
  // leader does the update
  if(lane_id() == leader)
    atomicAdd(ctr, __popc(mask));
}
